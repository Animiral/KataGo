#include "hip/hip_runtime.h"
#include "strengthnet.h"
#include <iostream>
#include <vector>

using namespace std;

namespace {

constexpr dim3 numBlocksForTensor(const Tensor& t, dim3 blockDim) noexcept {
  dim3 numBlocks(1, 1, 1);
  numBlocks.x = (t.dims.x + blockDim.x - 1) / blockDim.x;
  numBlocks.y = (t.dims.y + blockDim.y - 1) / blockDim.y;
  return numBlocks;
}

}

namespace StrengthNetKernels
{

__global__ void scale(Tensor y, const Tensor w) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i >= y.dims.x * y.dims.y)
    return;

  y.data[i] *= w.data[0];
}

__global__ void add(Tensor y, const Tensor a, const Tensor b) {
  assert(y.dims.x == a.dims.x);
  assert(y.dims.y == a.dims.y);
  assert(y.dims.x == b.dims.x);
  assert(y.dims.y == b.dims.y);

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i >= y.dims.x * y.dims.y)
    return;

  y.data[i] = a.data[i] + b.data[i];
}

__global__ void dotproduct(Tensor y, const Tensor a, const Tensor b) {
  assert(a.dims.x == b.dims.x);

  float s = 0;
  for(int i = 0; i < a.dims.x; i++)
    s += a.data[i] * b.data[i];
  y.data[0] = s;
}

// y = W*x
// set blocks to partition y into squares
__global__ void matmul(Tensor y, const Tensor W, const Tensor x) {
  assert(W.dims.x - x.dims.y <= 1); // weight.dims.x must either match x dims or have exactly 1 more column of bias weights
  assert(y.dims.x == x.dims.x);     // output size must match
  assert(y.dims.y == W.dims.y);     // output size must match

  // naive implementation
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // early exit for overspilling blocks
  if(row >= y.dims.y || col >= y.dims.x)
    return;

  size_t in_stride = x.dims.y;
  size_t w_stride = W.dims.y;
  size_t out_stride = y.dims.y;

  float h = 0.0f;
  for (int i = 0; i < x.dims.y; i++) {
    h += W.data[i * w_stride + row] * x.data[col * in_stride + i];
  }
  if(W.dims.x - x.dims.y > 0) // weight matrix includes bias row
    h += W.data[(w_stride - 1) * w_stride + row];

  y.data[col * out_stride + row] = h;
}

// y = a*b^T
// set blocks to partition y into squares
__global__ void transposeMatmul(Tensor y, const Tensor a, const Tensor b) {
  assert(a.dims.x == b.dims.x);  // input sizes must match
  assert(y.dims.x - b.dims.y <= 1);  // output size must either match or fit exactly 1 more column of bias weights
  assert(y.dims.y == a.dims.y);  // output size must match

  // naive implementation
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // early exit for overspilling blocks
  if(row >= y.dims.y || col >= y.dims.x)
    return;

  size_t b_stride = b.dims.y;
  size_t a_stride = a.dims.y;
  size_t y_stride = y.dims.y;

  float h = 0.0f;
  for (int i = 0; i < b.dims.x; i++) {
    if(col < b.dims.y)
      h += a.data[i * a_stride + row] * b.data[i * b_stride + col];
    else // construct bias row (as if b.data[...] == 1)
      h += a.data[i * a_stride + row];
  }
  y.data[col * y_stride + row] = h;
}

__global__ void relu(Tensor h) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i >= h.dims.x * h.dims.y)
    return;

  if(h.data[i] < 0)
    h.data[i] = 0;
}

__device__ float max(const Tensor& a) {
  extern __shared__ float buffer[];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  buffer[i] = a.data[i];
  __syncthreads();

  uint n = a.dims.x;
  for (uint s = 1; s < n; s *= 2) {
      if (i + s < n && buffer[i] < buffer[i + s]) {
          buffer[i] = buffer[i + s];
      }
      __syncthreads();
  }
  return buffer[0];
}

__device__ float sum(const Tensor& a) {
  extern __shared__ float buffer[];
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  buffer[i] = a.data[i];
  __syncthreads();

  uint n = a.dims.x;
  for (uint s = 1; s < n; s *= 2) {
      if (i + s < n) {
          buffer[i] += buffer[i + s];
      }
      __syncthreads();
  }
  return buffer[0];
}

// uses a.dims.x shared memory floats
__global__ void softmax(Tensor a) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i >= a.dims.x)
    return;

  a.data[i] = expf(a.data[i] - max(a));  // -max(a) improves numerical stability without changing the result
  a.data[i] /= sum(a);
}

__global__ void lossDerived(Tensor y_grad, float target, const Tensor y) {
  y_grad.data[0] = 2.f * (y.data[0] - target);
}

__global__ void softmaxDerived(Tensor z_grad, const Tensor a) {
  assert(z_grad.dims.x == a.dims.x);

  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if(j >= z_grad.dims.x)
    return;

  float a_j = a.data[j];
  float b = 0;
  for(uint i = 0; i < a.dims.x; i++) {
    float delta = i == threadIdx.x ? 1.f : 0.f;
    b += z_grad.data[i] * a.data[i] * (delta - a_j);
  }
  __syncthreads();
  z_grad.data[j] = b;
}

// set blocks to partition x_grad into squares
__global__ void matmulDerived(Tensor x_grad, const Tensor y_grad, const Tensor W) {
  assert(W.dims.x - x_grad.dims.y <= 1);  // weight.dims.x must either match x dims or have exactly 1 more column of bias weights
  assert(y_grad.dims.x == x_grad.dims.x); // output size must match
  assert(y_grad.dims.y == W.dims.y);      // output size must match

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // early exit for overspilling blocks
  if(row >= x_grad.dims.y || col >= x_grad.dims.x)
    return;

  size_t y_stride = y_grad.dims.y;
  size_t w_stride = W.dims.y;
  size_t x_stride = x_grad.dims.y;

  float h = 0.0f;
  for (int i = 0; i < y_grad.dims.y; i++) {
    h += W.data[row * w_stride + i] * y_grad.data[col * y_stride + i];
  }
  x_grad.data[col * x_stride + row] = h;
}

// 1 block with W.dims threads
__global__ void update(Tensor W, const Tensor W_grad, float learnrate) {
  assert(W.dims.x == W_grad.dims.x);
  assert(W.dims.y == W_grad.dims.y);

  W.data[threadIdx.x * W.dims.y + threadIdx.y] -= W_grad.data[threadIdx.x * W_grad.dims.y + threadIdx.y] * learnrate;
}

// __global__ void forwardTanhKernel(float* softx, int ch, int row) {
//   softx[threadIdx.x * ch + row] = 10.f * tanhf(softx[threadIdx.x * ch + row]);
// }

// __global__ void backwardTanhKernel(float* ingrads, float* outputx, float* outgrads) {
//   float cosa = cosf(outputx[threadIdx.x*2 + 1]);
//   outgrads[threadIdx.x*2 + 1] = ingrads[threadIdx.x*2 + 1] * 10.f / (cosa*cosa);
// }

} // end namespace StrengthNetKernels

using namespace StrengthNetKernels;

void StrengthNet::forward() {
  uint N = x.dims.x;
  dim3 blockDim1d(1024);
  dim3 blockDim2d(16, 16);

  // layer 1
  dim3 numBlocks = numBlocksForTensor(h, blockDim2d);
  matmul<<<numBlocks, blockDim2d>>>(h, W1, x);
  numBlocks = dim3((N * h.dims.y + N - 1) / N);
  relu<<<numBlocks, blockDim1d>>>(h);

  // layer 2
  numBlocks = numBlocksForTensor(r, blockDim2d);
  matmul<<<numBlocks, blockDim2d>>>(r, W2r, h);
  numBlocks = numBlocksForTensor(a, blockDim2d);
  matmul<<<numBlocks, blockDim2d>>>(a, W2z, h);

  // aggregate by attention
  numBlocks = numBlocksForTensor(a, blockDim1d);
  softmax<<<numBlocks, blockDim1d, N*sizeof(float)>>>(a);
  dotproduct<<<1, 1>>>(y, r, a);
}

void StrengthNet::backward(float target, float learnrate) {
  uint N = x.dims.x;
  dim3 blockDim1d(1024);
  dim3 blockDim2d(16, 16);

  target = (target - 1500.f) / 500.f;
  lossDerived<<<1,1>>>(y_grad, target, y); // dL/dy

  // aggregate by attention
  r_grad.assignFrom(a); // dy/dr
  dim3 numBlocks = numBlocksForTensor(r_grad, blockDim1d);
  scale<<<numBlocks, blockDim1d>>>(r_grad, y_grad); // dL/dr = dL/dy * dy/dr

  z_grad.assignFrom(r); // dy/da
  scale<<<numBlocks, blockDim1d>>>(z_grad, y_grad); // dL/da = dL/dy * dy/da
  softmaxDerived<<<numBlocks, blockDim1d>>>(z_grad, a); // dL/dz2 = da/dz2 * dL/da

  // layer 2
  numBlocks = numBlocksForTensor(W2r_grad, blockDim2d);
  transposeMatmul<<<numBlocks, blockDim2d>>>(W2r_grad, r_grad, h); // dL/dW2r = dL/dr * h^T
  numBlocks = numBlocksForTensor(W2z_grad, blockDim2d);
  transposeMatmul<<<numBlocks, blockDim2d>>>(W2z_grad, z_grad, h); // dL/dW2z = dL/dz * h^T

  numBlocks = numBlocksForTensor(hr_grad, blockDim2d);
  matmulDerived<<<numBlocks, blockDim2d>>>(hr_grad, r_grad, W2r);
  numBlocks = numBlocksForTensor(hz_grad, blockDim2d);
  matmulDerived<<<numBlocks, blockDim2d>>>(hz_grad, z_grad, W2z);
  numBlocks = dim3((N * h_grad.dims.y + N - 1) / N);
  add<<<numBlocks, blockDim1d>>>(h_grad, hr_grad, hz_grad); // dL/dh = dr/dh * dL/dr + dz/dh * dL/dz

  // layer 1
  relu<<<numBlocks, blockDim1d>>>(h_grad); // dL/dz1 = dL/dh * dh/dz1
 
  numBlocks = numBlocksForTensor(W1_grad, blockDim2d);
  transposeMatmul<<<numBlocks, blockDim2d>>>(W1_grad, h_grad, x); // dL/dW1 = dL/dz1 * x^T

  // apply gradients
  update<<<1, {W1.dims.x, W1.dims.y}>>>(W1, W1_grad, learnrate);
  update<<<1, {W2r.dims.x, W2r.dims.y}>>>(W2r, W2r_grad, learnrate);
  update<<<1, {W2z.dims.x, W2z.dims.y}>>>(W2z, W2z_grad, learnrate);
}

