#include "hip/hip_runtime.h"
#include "strengthnet.h"
#include <iostream>
#include <vector>
#include <cmath>

namespace {

constexpr dim3 numBlocksForTensor(const Tensor& t, dim3 blockDim);

}

namespace StrengthNetImpl
{

__device__ float& at(const Tensor& t, uint x, uint y = 0, uint z = 0); // tensor element access
__device__ float accumulate(float* data, uint n, float (*func)(float, float)); // Accumulate data with func

__global__ void scaleK(Tensor y, float w); // y = y * w
__global__ void hadamardK(Tensor y, const Tensor w); // y = y ⊙ w
__global__ void matmulK(Tensor y, const Tensor W, const Tensor x); // y = W * x
__global__ void addK(Tensor y, const Tensor x); // y = y + x
__global__ void minK(Tensor y, const Tensor x); // y = min(x) across x-dimension
__global__ void minDerivedK(Tensor x_grad, const Tensor y_grad, const Tensor x, const Tensor y); // x_grad = y_grad * d_min(x) / d_x
__global__ void sumK(Tensor y, const Tensor x); // y = sum(x) across x-dimension

__global__ void dotproduct(Tensor y, const Tensor a, const Tensor b); // y = a^T * b
__global__ void transposeMatmul(Tensor y, const Tensor a, const Tensor b, uint z_index); // y = a * b^T
__global__ void relu(Tensor h); // in-place relu
__global__ void softmax(Tensor a); // in-place softmax
__global__ void lossDerived(Tensor y_grad, float target, const Tensor y); // y_grad = d_y / d_target
__global__ void softmaxDerived(Tensor z_grad, const Tensor a); // z_grad = d_softmax(z) / d_z where a = softmax(z)
__global__ void matmulDerived(Tensor x_grad, const Tensor y_grad, const Tensor W); // x_grad = y_grad * d_y / d_x, where y = W * x
__global__ void accumulateTensorZ(Tensor W); // reduce z dimension by sum
__global__ void updateTensor(Tensor W, const Tensor W_grad, float weightPenalty, float learnrate); // W = W - W_grad * learnrate - d_(W ⊙ W) / d_W * weightPenalty

void scale(Tensor& y, float w);
void hadamard(Tensor& y, const Tensor& w);
void matmul(Tensor& y, const Tensor& W, const Tensor& x);
void add(Tensor& y, const Tensor& x);
void min(Tensor& y, const Tensor& x);
void minDerived(Tensor& x_grad, const Tensor& y_grad, const Tensor& x, const Tensor& y);
void sum(Tensor& y, const Tensor& x);

}

using namespace StrengthNetImpl;

void StrengthNet::forward() {
  // // layer 1
  matmul(h, W, x);
  b.broadcast(N, hidden_ch);
  add(h, b);
  b.broadcast(1, hidden_ch); // reset
  // matmul<<<numBlocks, blockDim2d>>>(h, W1, x);
  // numBlocks = dim3((N * h.dims.y + blockDim1d.x - 1) / blockDim1d.x);
  // relu<<<numBlocks, blockDim1d>>>(h);

  // // layer 2
  // numBlocks = numBlocksForTensor(r, blockDim2d);
  // matmul<<<numBlocks, blockDim2d>>>(r, W2r, h);
  // numBlocks = numBlocksForTensor(a, blockDim2d);
  // matmul<<<numBlocks, blockDim2d>>>(a, W2z, h);

  // // aggregate by attention
  min(y, h);
  // numBlocks = numBlocksForTensor(a, blockDim1d);
  // softmax<<<numBlocks, blockDim1d, N*sizeof(float)>>>(a);
  // dotproduct<<<1, 1>>>(y, r, a);
}

void StrengthNet::backward(float target/*, size_t index*/) {
  // assert(index < batchSize);

  target = (target - 1500.f) / 500.f;
  hipMemcpy(tgt.data, &target, sizeof(float), hipMemcpyHostToDevice);

  // dL/dy = 2(y - tgt)
  y_grad.assignFrom(y);
  scale(y_grad, 2.f);
  scale(tgt, -2.f);
  add(y_grad, tgt);
  // lossDerived<<<1,1>>>(y_grad, target, y); // dL/dy

  // dL/dh = dL/dy * I_min(h)
  minDerived(h_grad, y_grad, h, y);

  // dL/dW = dL/dh * x^T; dL/db = sum(dL/dh)
  x.transpose();
  matmul(W_grad, h_grad, x);
  x.transpose(); // reset
  sum(b_grad, h_grad);

  // // aggregate by attention
  // r_grad.assignFrom(a); // dy/dr
  // y_grad.broadcast(N);
  // hadamard(r_grad, y_grad); // dL/dr = dL/dy * dy/da

  // z_grad.assignFrom(r); // dy/da
  // hadamard(z_grad, y_grad); // dL/da = dL/dy * dy/da
  // softmaxDerived<<<numBlocks, blockDim1d>>>(z_grad, a); // dL/dz2 = da/dz2 * dL/da

  // // layer 2
  // numBlocks = numBlocksForTensor(W2r_grad, blockDim2d);
  // transposeMatmul<<<numBlocks, blockDim2d>>>(W2r_grad, r_grad, h, index); // dL/dW2r = dL/dr * h^T
  // transposeMatmul<<<numBlocks, blockDim2d>>>(W2z_grad, z_grad, h, index); // dL/dW2z = dL/dz * h^T

  // numBlocks = numBlocksForTensor(hr_grad, blockDim2d);
  // matmulDerived<<<numBlocks, blockDim2d>>>(hr_grad, r_grad, W2r);
  // matmulDerived<<<numBlocks, blockDim2d>>>(hz_grad, z_grad, W2z);
  // numBlocks = dim3((N * h_grad.dims.y + blockDim1d.x - 1) / blockDim1d.x);
  // add<<<numBlocks, blockDim1d>>>(h_grad, hr_grad, hz_grad); // dL/dh = dr/dh * dL/dr + dz/dh * dL/dz

  // // layer 1
  // relu<<<numBlocks, blockDim1d>>>(h_grad); // dL/dz1 = dL/dh * dh/dz1
 
  // numBlocks = numBlocksForTensor(W1_grad, blockDim2d);
  // transposeMatmul<<<numBlocks, blockDim2d>>>(W1_grad, h_grad, x, index); // dL/dW1 = dL/dz1 * x^T
}

void StrengthNet::mergeGrads() {
  accumulateTensorZ<<<1, {W_grad.dims.x, W_grad.dims.y}>>>(W_grad);
  accumulateTensorZ<<<1, {b_grad.dims.x, b_grad.dims.y}>>>(b_grad);
  W_grad.dims.z = b_grad.dims.z = 1;
  // accumulateTensorZ<<<1, {W1_grad.dims.x, W1_grad.dims.y}>>>(W1_grad);
  // accumulateTensorZ<<<1, {W2r_grad.dims.x, W2r_grad.dims.y}>>>(W2r_grad);
  // accumulateTensorZ<<<1, {W2z_grad.dims.x, W2z_grad.dims.y}>>>(W2z_grad);
  // W1_grad.dims.z = W2r_grad.dims.z = W2z_grad.dims.z = 1;
}

void StrengthNet::update(float weightPenalty, float learnrate) {
  updateTensor<<<1, {W.dims.x, W.dims.y}>>>(W, W_grad, weightPenalty, learnrate);
  updateTensor<<<1, {b.dims.x, b.dims.y}>>>(b, b_grad, weightPenalty, learnrate);
  // updateTensor<<<1, {W1.dims.x, W1.dims.y}>>>(W1, W1_grad, weightPenalty, learnrate);
  // updateTensor<<<1, {W2r.dims.x, W2r.dims.y}>>>(W2r, W2r_grad, weightPenalty, learnrate);
  // updateTensor<<<1, {W2z.dims.x, W2z.dims.y}>>>(W2z, W2z_grad, weightPenalty, learnrate);
}

namespace {

constexpr dim3 numBlocksForTensor(const Tensor& t, dim3 blockDim) {
  dim3 numBlocks(1, 1, 1);
  numBlocks.x = (t.dims.x + blockDim.x - 1) / blockDim.x;
  numBlocks.y = (t.dims.y + blockDim.y - 1) / blockDim.y;
  return numBlocks;
}

}

namespace StrengthNetImpl
{

__device__ float& at(const Tensor& t, uint x, uint y, uint z) {
  assert(x < t.viewDims.x);
  assert(y < t.viewDims.y);
  assert(z < t.viewDims.z);

  // implement broadcast
  if(1 == t.dims.x)
    x = 0;
  if(1 == t.dims.y)
    y = 0;
  if(1 == t.dims.z)
    z = 0;

  uint xy;

  // implement transposition
  if(t.transposed)
    xy = y * t.dims.x + x;
  else
    xy = x * t.dims.y + y;
  
  return t.data[z * t.dims.x * t.dims.y + xy];
}

__device__ float accumulate(float* data, uint n, float (*func)(float, float)) {
  assert(0 == blockIdx.x); // this operation cannot use blocks
  assert(blockDim.x >= n); // must have enough threads
  extern __shared__ float buffer[];
  uint i = threadIdx.x;

  if(i < n)
    buffer[i] = data[i];

  __syncthreads();

  for (uint s = n/2; s > 0; s = n/2) {
      if (i + n - s < n) { // accumulate second half of data into first half
          buffer[i] = func(buffer[i], buffer[i + n - s]);
      }
      n -= s; // discard second half
      __syncthreads();
  }

  // now we are down to exactly either zero, one or two elements
  if(2 == n)
    return func(buffer[0], buffer[1]);
  else if(1 == n)
    return buffer[0];
  else
    return NAN; // some default
}

__global__ void scaleK(Tensor y, float w) {
  uint xx = blockIdx.x * blockDim.x + threadIdx.x;
  uint yy = blockIdx.y * blockDim.y + threadIdx.y;
  uint zz = blockIdx.z * blockDim.z + threadIdx.z;
  if(xx < y.dims.x && yy < y.dims.y && zz < y.dims.z)
    at(y, xx, yy, zz) *= w;
}

__global__ void hadamardK(Tensor y, const Tensor w) {
  assert(y.dims.x == w.viewDims.x);
  assert(y.dims.y == w.viewDims.y);
  assert(y.dims.z == w.viewDims.z);
  uint xx = blockIdx.x * blockDim.x + threadIdx.x;
  uint yy = blockIdx.y * blockDim.y + threadIdx.y;
  uint zz = blockIdx.z * blockDim.z + threadIdx.z;
  if(xx < y.dims.x && yy < y.dims.y && zz < y.dims.z)
    at(y, xx, yy, zz) *= at(w, xx, yy, zz);
}

// y = W*x
// set blocks to partition y into squares
__global__ void matmulK(Tensor y, const Tensor W, const Tensor x) {
  assert(W.dims.x == x.dims.y); // input size must match
  assert(y.dims.x == x.dims.x); // output size must match
  assert(y.dims.y == W.dims.y); // output size must match
  assert(1 == W.dims.z);
  assert(1 == x.dims.z); // TODO: support parallelism

  // naive implementation
  uint row = blockIdx.y * blockDim.y + threadIdx.y;
  uint col = blockIdx.x * blockDim.x + threadIdx.x;

  // early exit for overspilling blocks
  if(row >= y.dims.y || col >= y.dims.x)
    return;

  float h = 0.0f;
  for (uint i = 0; i < x.dims.y; i++) {
    h += at(W, i, row) * at(x, col, i);
  }
  at(y, col, row) = h;
}

__global__ void addK(Tensor y, const Tensor x) {
  assert(y.dims.x == x.viewDims.x);
  assert(y.dims.y == x.viewDims.y);
  assert(y.dims.z == x.viewDims.z);

  uint i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i >= y.dims.x * y.dims.y)
    return;

  y.data[i] += x.data[i];
}

__global__ void minK(Tensor y, const Tensor x) {
  assert(1 == y.dims.x);
  assert(y.dims.y == x.viewDims.y);
  assert(y.dims.z == x.viewDims.z);
  assert(1 == y.dims.y); // TODO: parallelism support
  assert(1 == y.dims.z); // TODO: parallelism support

  // uint yy = blockIdx.y * blockDim.y + threadIdx.y;
  // uint zz = blockIdx.z * blockDim.z + threadIdx.z;
  // if(yy >= y.dims.y || zz >= y.dims.z)
  //   return;

  float minValue = accumulate(x.data, x.dims.x, [](float a, float b) { return a < b ? a : b; });
  if(0 == threadIdx.x)
    y.data[0] = minValue;
}

__global__ void minDerivedK(Tensor x_grad, const Tensor y_grad, const Tensor x, const Tensor y) {
  assert(x_grad.dims.x == x.viewDims.x);
  assert(x_grad.dims.y == x.viewDims.y);
  assert(x_grad.dims.z == x.viewDims.z);
  assert(y_grad.viewDims.x == y.viewDims.x);
  assert(y_grad.viewDims.y == y.viewDims.y);
  assert(y_grad.viewDims.z == y.viewDims.z);
  assert(1 == y.viewDims.x);
  assert(y.viewDims.y == x.viewDims.y);
  assert(y.viewDims.z == x.viewDims.z);
  assert(1 == y.viewDims.y); // TODO: parallelism support
  assert(1 == y.viewDims.z); // TODO: parallelism support
  assert(0 == blockIdx.x); // TODO: parallelism support

  if(threadIdx.x >= x_grad.dims.x)
    return;

  float minValue = y.data[0];
  x_grad.data[threadIdx.x] = y_grad.data[0] * (x.data[threadIdx.x] == minValue);
}

__global__ void sumK(Tensor y, const Tensor x) {
  assert(1 == y.dims.x);
  assert(y.dims.y == x.viewDims.y);
  assert(y.dims.z == x.viewDims.z);
  assert(1 == y.dims.y); // TODO: parallelism support
  assert(1 == y.dims.z); // TODO: parallelism support

  float sumValue = accumulate(x.data, x.viewDims.x, [](float a, float b) { return a + b; });
  if(0 == threadIdx.x)
    y.data[0] = sumValue;
}

__global__ void dotproduct(Tensor y, const Tensor a, const Tensor b) {
  assert(a.dims.x == b.dims.x);

  float s = 0;
  for(int i = 0; i < a.dims.x; i++)
    s += at(a, i) * at(b, i);
  at(y, 0) = s;
}

// y[z_index] = a*b^T
// set blocks to partition y into squares
__global__ void transposeMatmul(Tensor y, const Tensor a, const Tensor b, uint z_index) {
  assert(a.dims.x == b.dims.x);  // input sizes must match
  assert(y.dims.x - b.dims.y <= 1);  // output size must either match or fit exactly 1 more column of bias weights
  assert(y.dims.y == a.dims.y);  // output size must match
  assert(z_index < y.dims.z); // must have room to store the result

  // naive implementation
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // early exit for overspilling blocks
  if(row >= y.dims.y || col >= y.dims.x)
    return;

  float h = 0.0f;
  for (int i = 0; i < b.dims.x; i++) {
    if(col < b.dims.y)
      h += at(a, i, row) * at(b, i, col);
    else // construct bias row (as if b.data[...] == 1)
      h += at(a, i, row);
  }
  at(y, col, row, z_index) = h;
}

__global__ void relu(Tensor h) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i >= h.dims.x * h.dims.y)
    return;

  if(h.data[i] < 0)
    h.data[i] = 0;
}

// uses a.dims.x shared memory floats
__global__ void softmax(Tensor a) {
  assert(1 == a.dims.y);
  assert(1 == a.dims.z); // TODO: batch softmax
  assert(a.dims.x == a.viewDims.x);

  int i = threadIdx.x;
  if(i >= a.dims.x)
    return;

  float max_a = accumulate(a.data, a.dims.x, [](float a, float b) { return a > b ? a : b; });
  at(a, i) = expf(at(a, i) - max_a);  // -max(a) improves numerical stability without changing the result
  float sum_a = accumulate(a.data, a.dims.x, [](float a, float b) { return a + b; });
  at(a, i) /= sum_a;
}

__global__ void lossDerived(Tensor y_grad, float target, const Tensor y) {
  at(y_grad, 0) = 2.f * (at(y, 0) - target);
}

__global__ void softmaxDerived(Tensor z_grad, const Tensor a) {
  assert(z_grad.dims.x == a.dims.x);

  int j = blockIdx.x * blockDim.x + threadIdx.x;
  if(j >= z_grad.dims.x)
    return;

  float a_j = at(a, j);
  float b = 0;
  for(uint i = 0; i < a.dims.x; i++) {
    float delta = i == threadIdx.x ? 1.f : 0.f;
    b += at(z_grad, i) * at(a, i) * (delta - a_j);
  }
  __syncthreads();
  at(z_grad, j) = b;
}

// set blocks to partition x_grad into squares
__global__ void matmulDerived(Tensor x_grad, const Tensor y_grad, const Tensor W) {
  assert(W.dims.x - x_grad.dims.y <= 1);  // weight.dims.x must either match x dims or have exactly 1 more column of bias weights
  assert(y_grad.dims.x == x_grad.dims.x); // output size must match
  assert(y_grad.dims.y == W.dims.y);      // output size must match

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // early exit for overspilling blocks
  if(row >= x_grad.dims.y || col >= x_grad.dims.x)
    return;

  float h = 0.0f;
  for (int i = 0; i < y_grad.dims.y; i++) {
    h += at(W, row, i) * at(y_grad, col, i);
  }
  at(x_grad, col, row) = h;
}

// 1 block with W.dims threads
__global__ void accumulateTensorZ(Tensor W) {
  float v = 0;
  for(uint z = 0; z < W.dims.z; z++) {
    v += at(W, threadIdx.x, threadIdx.y, z);
  }
  at(W, threadIdx.x, threadIdx.y, 0) = v;
}

// 1 block with W.dims threads
__global__ void updateTensor(Tensor W, const Tensor W_grad, float weightPenalty, float learnrate) {
  assert(W.dims.x == W_grad.dims.x);
  assert(W.dims.y == W_grad.dims.y);
  assert(1 == W_grad.dims.z);

  float delta = 0;
  for(uint z = 0; z < W_grad.dims.z; z++) {
    delta += at(W_grad, threadIdx.x, threadIdx.y, z);
  }
  delta += at(W, threadIdx.x, threadIdx.y) * 2 * weightPenalty;
  at(W, threadIdx.x, threadIdx.y) -= delta * learnrate;
}

// __global__ void forwardTanhKernel(float* softx, int ch, int row) {
//   softx[threadIdx.x * ch + row] = 10.f * tanhf(softx[threadIdx.x * ch + row]);
// }

// __global__ void backwardTanhKernel(float* ingrads, float* outputx, float* outgrads) {
//   float cosa = cosf(outputx[threadIdx.x*2 + 1]);
//   outgrads[threadIdx.x*2 + 1] = ingrads[threadIdx.x*2 + 1] * 10.f / (cosa*cosa);
// }

void scale(Tensor& y, float w) {
  dim3 blockDim(16, 16, 4);
  dim3 numBlocks = numBlocksForTensor(y, blockDim);
  scaleK<<<numBlocks, blockDim>>>(y, w);
}

void hadamard(Tensor& y, const Tensor& w) {
  dim3 blockDim(16, 16, 4);
  dim3 numBlocks = numBlocksForTensor(y, blockDim);
  hadamardK<<<numBlocks, blockDim>>>(y, w);
}

void matmul(Tensor& y, const Tensor& W, const Tensor& x) {
  dim3 blockDim(16, 16);
  dim3 numBlocks = numBlocksForTensor(y, blockDim);
  matmulK<<<numBlocks, blockDim>>>(y, W, x);
}

void add(Tensor& y, const Tensor& x) {
  uint numBlocks = (y.dims.x * y.dims.y * y.dims.z + 1023) / 1024;
  addK<<<numBlocks, 1024>>>(y, x);
}

void min(Tensor& y, const Tensor& x) {
  uint numBlocks = (x.dims.x + 1023) / 1024;
  minK<<<numBlocks, 1024, x.dims.x*sizeof(float)>>>(y, x);
}

void minDerived(Tensor& x_grad, const Tensor& y_grad, const Tensor& x, const Tensor& y) {
  uint numBlocks = (x.dims.x + 1023) / 1024;
  minDerivedK<<<numBlocks, 1024, x.dims.x*sizeof(float)>>>(x_grad, y_grad, x, y);
}

void sum(Tensor& y, const Tensor& x) {
  uint numBlocks = (x.dims.x + 1023) / 1024;
  sumK<<<numBlocks, 1024, x.dims.x*sizeof(float)>>>(y, x);
}

} // end namespace StrengthNetImpl
